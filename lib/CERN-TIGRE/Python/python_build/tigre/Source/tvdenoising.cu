#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * MATLAB MEX  functions for TV image denoising. Check inputs and parses 
 * MATLAB data to C++ data.
 *
 *
 * CODE by   Imanol Luengo
 *           PhD student University of Nottingham
 *           imaluengo@gmail.com
 *           2015
 *           Sligtly modified by Ander Biguri
---------------------------------------------------------------------------
---------------------------------------------------------------------------
Copyright (c) 2015, University of Bath and CERN- European Organization for 
Nuclear Research
All rights reserved.

Redistribution and use in source and binary forms, with or without 
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, 
this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice, 
this list of conditions and the following disclaimer in the documentation 
and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors
may be used to endorse or promote products derived from this software without
specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" 
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE 
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE 
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE 
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR 
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF 
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN 
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.
 ---------------------------------------------------------------------------

Contact: tigre.toolbox@gmail.com
Codes  : https://github.com/CERN/TIGRE
--------------------------------------------------------------------------- 
 */



// http://gpu4vision.icg.tugraz.at/papers/2010/knoll.pdf#pub47
#define MAXTREADS 1024

#include "tvdenoising.hpp"
#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                mexErrMsgIdAndTxt("CBCT:CUDA:TVdenoising",hipGetErrorString(__err));\
        } \
} while (0)



__device__ __inline__
float divergence(const float* pz, const float* py, const float* px,
                 long z, long y, long x, long depth, long rows, long cols,
                 float dz, float dy, float dx)
{
    long size2d = rows*cols;
    long idx = z * size2d + y * cols + x;
    float _div = 0.0f;

    if ( z - 1 >= 0 ) {
        _div += (pz[idx] - pz[(z-1)*size2d + y*cols + x]) / dz;
    } else {
        _div += pz[idx];
    }

    if ( y - 1 >= 0 ) {
        _div += (py[idx] - py[z*size2d + (y-1)*cols + x]) / dy;
    } else {
        _div += py[idx];
    }

    if ( x - 1 >= 0 ) {
        _div += (px[idx] - px[z*size2d + y*cols + (x-1)]) / dx;
    } else {
        _div += px[idx];
    }

    return _div;
}

__device__ __inline__
void gradient(const float* u, float* grad,
              long z, long y, long x,
              long depth, long rows, long cols,
              float dz, float dy, float dx)
{
    long size2d = rows*cols;
    long idx = z * size2d + y * cols + x;

    float uidx = u[idx];

    if ( z + 1 < depth ) {
        grad[0] = (u[(z+1)*size2d + y*cols + x] - uidx) / dz;
    }

    if ( y + 1 < rows ) {
        grad[1] = (u[z*size2d + (y+1)*cols + x] - uidx) / dy;
    }

    if ( x + 1 < cols ) {
        grad[2] = (u[z*size2d + y*cols + (x+1)] - uidx) / dx;
    }
}


__global__
void update_u(const float* f, const float* pz, const float* py, const float* px, float* u,
              float tau, float lambda,
              long depth, long rows, long cols,
              float dz, float dy, float dx)
{
    long x = threadIdx.x + blockIdx.x * blockDim.x;
    long y = threadIdx.y + blockIdx.y * blockDim.y;
    long z = threadIdx.z + blockIdx.z * blockDim.z;
    long idx = z * rows * cols + y * cols + x;

    if ( x >= cols || y >= rows || z >= depth )
        return;

    float _div = divergence(pz, py, px, z, y, x, depth, rows, cols, dz, dy, dx);

    u[idx] = u[idx] * (1.0f - tau) + tau * (f[idx] + (1.0f/lambda) * _div);
}


__global__
void update_p(const float* u, float* pz, float* py, float* px,
              float tau, long depth, long rows, long cols,
              float dz, float dy, float dx)
{
    long x = threadIdx.x + blockIdx.x * blockDim.x;
    long y = threadIdx.y + blockIdx.y * blockDim.y;
    long z = threadIdx.z + blockIdx.z * blockDim.z;
    long idx = z * rows * cols + y * cols + x;

    if ( x >= cols || y >= rows || z >= depth )
        return;

    float grad[3] = {0,0,0}, q[3];
    gradient(u, grad, z, y, x, depth, rows, cols, dz, dy, dx);

    q[0] = pz[idx] + tau * grad[0];
    q[1] = py[idx] + tau * grad[1];
    q[2] = px[idx] + tau * grad[2];

    float norm = fmaxf(1.0f, sqrtf(q[0] * q[0] + q[1] * q[1] + q[2] * q[2]));

    pz[idx] = q[0] / norm;
    py[idx] = q[1] / norm;
    px[idx] = q[2] / norm;
}


// Main function
void tvdenoising(const float* src, float* dst, float lambda,
                 const float* spacing, const long* image_size, int maxIter)
{
    // Init params
    size_t total_pixels = image_size[0] * image_size[1]  * image_size[2] ;
    size_t mem_size = sizeof(float) * total_pixels;



    float *d_src, *d_u, *d_px, *d_py, *d_pz;

    // F
    hipMalloc(&d_src, mem_size);
    hipMemcpy(d_src, src, mem_size, hipMemcpyHostToDevice);
    cudaCheckErrors("Memory Malloc and Memset: SRC");

    

    // U
    hipMalloc(&d_u, mem_size);
    hipMemcpy(d_u, d_src, mem_size, hipMemcpyDeviceToDevice);
    cudaCheckErrors("Memory Malloc and Memset: U");
    // PX
    hipMalloc(&d_px, mem_size);
    hipMemset(d_px, 0, mem_size);
    cudaCheckErrors("Memory Malloc and Memset: PX");
    // PY
    hipMalloc(&d_py, mem_size);
    hipMemset(d_py, 0, mem_size);
    cudaCheckErrors("Memory Malloc and Memset: PY");
    // PZ
    hipMalloc(&d_pz, mem_size);
    hipMemset(d_pz, 0, mem_size);
    cudaCheckErrors("Memory Malloc and Memset: PZ");

    // bdim and gdim
    dim3 block(10, 10, 10);
    dim3 grid((image_size[0]+block.x-1)/block.x, (image_size[1]+block.y-1)/block.y, (image_size[2]+block.z-1)/block.z);

    int i = 0;

    float tau2, tau1;
    
    for ( i = 0; i < maxIter; i++ )
    {
        tau2 = 0.3f + 0.02f * i;
        tau1 = (1.f/tau2) * ((1.f/6.f) - (5.f/(15.f+i)));

        update_u<<<grid, block>>>(d_src, d_pz, d_py, d_px, d_u, tau1, lambda,
                                  image_size[2], image_size[1],image_size[0],
                                  spacing[2], spacing[1], spacing[0]);

        update_p<<<grid, block>>>(d_u, d_pz, d_py, d_px, tau2,
                                  image_size[2], image_size[1], image_size[0],
                                  spacing[2], spacing[1], spacing[0]);
        
    }

    cudaCheckErrors("TV minimization"); 

    hipMemcpy(dst, d_u, mem_size, hipMemcpyDeviceToHost);
    cudaCheckErrors("Copy result back");

    hipFree(d_src);
    hipFree(d_u);
    hipFree(d_pz);
    hipFree(d_py);
    hipFree(d_px);
    //hipDeviceReset();
}