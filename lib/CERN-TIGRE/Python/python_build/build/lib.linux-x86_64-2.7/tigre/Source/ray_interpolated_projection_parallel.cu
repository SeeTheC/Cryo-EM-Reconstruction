#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA functions for texture-memory interpolation based projection
 *
 * This file has the necesary functions to perform X-ray parallel projection 
 * operation given a geaometry, angles and image. It uses the 3D texture 
 * memory linear interpolation to uniformily sample a path to integrate the 
 * X-rays.
 *
 * CODE by       Ander Biguri
 *
---------------------------------------------------------------------------
---------------------------------------------------------------------------
Copyright (c) 2015, University of Bath and CERN- European Organization for 
Nuclear Research
All rights reserved.

Redistribution and use in source and binary forms, with or without 
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, 
this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice, 
this list of conditions and the following disclaimer in the documentation 
and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors
may be used to endorse or promote products derived from this software without
specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" 
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE 
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE 
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE 
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR 
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF 
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN 
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.
 ---------------------------------------------------------------------------

Contact: tigre.toolbox@gmail.com
Codes  : https://github.com/CERN/TIGRE
--------------------------------------------------------------------------- 
 */



#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "ray_interpolated_projection_parallel.hpp"
//#include "mex.h"
#include <math.h>


//        if (__err != hipSuccess) { \
//                  printf("%s \n", msg);\
//                  printf("%s \n", hipGetErrorString(__err));\
//        } \
// TODO: Error logging
#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
} while (0)
    
    
// Declare the texture reference.
    texture<float, hipTextureType3D , hipReadModeElementType> tex;

#define MAXTREADS 1024
/*GEOMETRY DEFINITION
 *
 *                Detector plane, behind
 *            |-----------------------------|
 *            |                             |
 *            |                             |
 *            |                             |
 *            |                             |
 *            |      +--------+             |
 *            |     /        /|             |
 *   A Z      |    /        / |*D           |
 *   |        |   +--------+  |             |
 *   |        |   |        |  |             |
 *   |        |   |     *O |  +             |
 *    --->y   |   |        | /              |
 *  /         |   |        |/               |
 * V X        |   +--------+                |
 *            |-----------------------------|
 *
 *           *S
 *
 *
 *
 *
 *
 **/


__global__ void kernelPixelDetector_parallel( Geometry geo,
        float* detector,
        Point3D source ,
        Point3D deltaU,
        Point3D deltaV,
        Point3D uvOrigin,
        float maxdist){
    
    unsigned long y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned long x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long idx =  x  * geo.nDetecV + y;

    if ((x>= geo.nDetecU) | (y>= geo.nDetecV))
        return;
    
    

    
    /////// Get coordinates XYZ of pixel UV
    int pixelV = geo.nDetecV-y-1;
    int pixelU = x;
    
    
    
    float vectX,vectY,vectZ;
    Point3D P;
    P.x=(uvOrigin.x+pixelU*deltaU.x+pixelV*deltaV.x);
    P.y=(uvOrigin.y+pixelU*deltaU.y+pixelV*deltaV.y);
    P.z=(uvOrigin.z+pixelU*deltaU.z+pixelV*deltaV.z);
    Point3D S;
    S.x=(source.x+pixelU*deltaU.x+pixelV*deltaV.x);
    S.y=(source.y+pixelU*deltaU.y+pixelV*deltaV.y);
    S.z=(source.z+pixelU*deltaU.z+pixelV*deltaV.z);
    
    // Length is the ray length in normalized space
    double length=sqrt((S.x-P.x)*(S.x-P.x)+(S.y-P.y)*(S.y-P.y)+(S.z-P.z)*(S.z-P.z));
    //now legth is an integer of Nsamples that are required on this line
    length=ceil(length/geo.accuracy);//Divide the directional vector by an integer
    vectX=(P.x -S.x)/(length);
    vectY=(P.y -S.y)/(length);
    vectZ=(P.z -S.z)/(length);
    
    
//     //Integrate over the line
    float tx,ty,tz;
    float sum=0;
    float i;
    
    
    // limit the amount of mem access after the cube, but before the detector.
    if ((geo.DSO/geo.dVoxelX+maxdist)/geo.accuracy  <   length)
        length=ceil((geo.DSO/geo.dVoxelX+maxdist)/geo.accuracy);  
    //Length is not actually a length, but the amount of memreads with given accuracy ("samples per voxel")
    
    for (i=floor(maxdist/geo.accuracy); i<=length; i=i+1){
        tx=vectX*i+S.x;
        ty=vectY*i+S.y;
        tz=vectZ*i+S.z;
        
        sum += tex3D(tex, tx+0.5, ty+0.5, tz+0.5); // this line is 94% of time.
        
    }
    float deltalength=sqrt((vectX*geo.dVoxelX)*(vectX*geo.dVoxelX)+
            (vectY*geo.dVoxelY)*(vectY*geo.dVoxelY)+(vectZ*geo.dVoxelZ)*(vectZ*geo.dVoxelZ) );
    detector[idx]=sum*deltalength;
}



int interpolation_projection_parallel(float const * const img, Geometry geo, float** result,float const * const alphas,int nalpha){
    
 
    // copy data to CUDA memory

    hipArray *d_imagedata = 0;
    
    const hipExtent extent = make_hipExtent(geo.nVoxelX, geo.nVoxelY, geo.nVoxelZ);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipMalloc3DArray(&d_imagedata, &channelDesc, extent);
    cudaCheckErrors("hipMalloc3D error 3D tex");
    
    hipMemcpy3DParms copyParams = { 0 };
    copyParams.srcPtr = make_hipPitchedPtr((void*)img, extent.width*sizeof(float), extent.width, extent.height);
    copyParams.dstArray = d_imagedata;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);
    
    cudaCheckErrors("hipMemcpy3D fail");
    
    // Configure texture options
    tex.normalized = false;
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeBorder;
    tex.addressMode[1] = hipAddressModeBorder;
    tex.addressMode[2] = hipAddressModeBorder;
    
    hipBindTextureToArray(tex, d_imagedata, channelDesc);
    
    cudaCheckErrors("3D texture memory bind fail");
    
    
    //Done! Image put into texture memory.
    
    
    size_t num_bytes = geo.nDetecU*geo.nDetecV * sizeof(float);
    float* dProjection;
    hipMalloc((void**)&dProjection, num_bytes);
    cudaCheckErrors("hipMalloc fail");

    
//     If we are going to time
    bool timekernel=false;
    hipEvent_t start, stop;
    float elapsedTime;
    if (timekernel){
        hipEventCreate(&start);
        hipEventRecord(start,0);
    } 
    
    // 16x16 gave the best performance empirically
    // Funnily that makes it compatible with most GPUs.....
    dim3 grid(ceil((float)geo.nDetecU/32),ceil((float)geo.nDetecV/32),1);
    dim3 block(32,32,1); 
    Point3D source, deltaU, deltaV, uvOrigin;
    float maxdist;
    for (unsigned int i=0;i<nalpha;i++){
        
        geo.alpha=alphas[i];
        //precomute distances for faster execution
        maxdist=maxDistanceCubeXY(geo,geo.alpha,i);
        //Precompute per angle constant stuff for speed
        computeDeltas_parallel(geo,geo.alpha,i, &uvOrigin, &deltaU, &deltaV, &source);
        //Interpolation!!
        
        kernelPixelDetector_parallel<<<grid,block>>>(geo,dProjection, source, deltaU, deltaV, uvOrigin,floor(maxdist));
        cudaCheckErrors("Kernel fail");
        // copy result to host
        hipMemcpy(result[i], dProjection, num_bytes, hipMemcpyDeviceToHost);
        cudaCheckErrors("hipMemcpy fail");
        
           

    }
    if (timekernel){
        hipEventCreate(&stop);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start,stop);
        //TODO: replace this
//        mexPrintf("%f\n" ,elapsedTime);
    }

    hipUnbindTexture(tex);
    cudaCheckErrors("Unbind  fail");
    
    hipFree(dProjection);
    hipFreeArray(d_imagedata);
    cudaCheckErrors("hipFree d_imagedata fail");
    
    
    
    //hipDeviceReset();
    
    return 0;
}




/* This code precomputes The location of the source and the Delta U and delta V (in the warped space)
 * to compute the locations of the x-rays. While it seems verbose and overly-optimized,
 * it does saves about 30% of each of the kernel calls. Thats something!
 **/
void computeDeltas_parallel(Geometry geo, float alpha,int i, Point3D* uvorigin, Point3D* deltaU, Point3D* deltaV, Point3D* source){
    Point3D S;
    S.x=geo.DSO;
    S.y=geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);
    S.z=geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    
    //End point
    Point3D P,Pu0,Pv0;
    
    P.x  =-(geo.DSD-geo.DSO);   P.y  = geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       P.z  = geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pu0.x=-(geo.DSD-geo.DSO);   Pu0.y= geo.dDetecU*(1-((float)geo.nDetecU/2)+0.5);       Pu0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pv0.x=-(geo.DSD-geo.DSO);   Pv0.y= geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       Pv0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-1);
    // Geomtric trasnformations:
    
    //1: Offset detector
    
    //P.x
    P.y  =P.y  +geo.offDetecU[i];    P.z  =P.z  +geo.offDetecV[i];
    Pu0.y=Pu0.y+geo.offDetecU[i];    Pu0.z=Pu0.z+geo.offDetecV[i];
    Pv0.y=Pv0.y+geo.offDetecU[i];    Pv0.z=Pv0.z+geo.offDetecV[i];
    //S doesnt need to chagne
    
    
    //3: Rotate (around z)!
    Point3D Pfinal, Pfinalu0, Pfinalv0;
    
    Pfinal.x  =P.x*cos(geo.alpha)-P.y*sin(geo.alpha);       Pfinal.y  =P.y*cos(geo.alpha)+P.x*sin(geo.alpha);       Pfinal.z  =P.z;
    Pfinalu0.x=Pu0.x*cos(geo.alpha)-Pu0.y*sin(geo.alpha);   Pfinalu0.y=Pu0.y*cos(geo.alpha)+Pu0.x*sin(geo.alpha);   Pfinalu0.z=Pu0.z;
    Pfinalv0.x=Pv0.x*cos(geo.alpha)-Pv0.y*sin(geo.alpha);   Pfinalv0.y=Pv0.y*cos(geo.alpha)+Pv0.x*sin(geo.alpha);   Pfinalv0.z=Pv0.z;
    
    Point3D S2;
    S2.x=S.x*cos(geo.alpha)-S.y*sin(geo.alpha);
    S2.y=S.y*cos(geo.alpha)+S.x*sin(geo.alpha);
    S2.z=S.z;
    
    //2: Offset image (instead of offseting image, -offset everything else)
    
    Pfinal.x  =Pfinal.x-geo.offOrigX[i];     Pfinal.y  =Pfinal.y-geo.offOrigY[i];     Pfinal.z  =Pfinal.z-geo.offOrigZ[i];
    Pfinalu0.x=Pfinalu0.x-geo.offOrigX[i];   Pfinalu0.y=Pfinalu0.y-geo.offOrigY[i];   Pfinalu0.z=Pfinalu0.z-geo.offOrigZ[i];
    Pfinalv0.x=Pfinalv0.x-geo.offOrigX[i];   Pfinalv0.y=Pfinalv0.y-geo.offOrigY[i];   Pfinalv0.z=Pfinalv0.z-geo.offOrigZ[i];
    S2.x=S2.x-geo.offOrigX[i];       S2.y=S2.y-geo.offOrigY[i];       S2.z=S2.z-geo.offOrigZ[i];
    
    // As we want the (0,0,0) to be in a corner of the image, we need to translate everything (after rotation);
    Pfinal.x  =Pfinal.x+geo.sVoxelX/2-geo.dVoxelX/2;      Pfinal.y  =Pfinal.y+geo.sVoxelY/2-geo.dVoxelY/2;          Pfinal.z  =Pfinal.z  +geo.sVoxelZ/2-geo.dVoxelZ/2;
    Pfinalu0.x=Pfinalu0.x+geo.sVoxelX/2-geo.dVoxelX/2;    Pfinalu0.y=Pfinalu0.y+geo.sVoxelY/2-geo.dVoxelY/2;        Pfinalu0.z=Pfinalu0.z+geo.sVoxelZ/2-geo.dVoxelZ/2;
    Pfinalv0.x=Pfinalv0.x+geo.sVoxelX/2-geo.dVoxelX/2;    Pfinalv0.y=Pfinalv0.y+geo.sVoxelY/2-geo.dVoxelY/2;        Pfinalv0.z=Pfinalv0.z+geo.sVoxelZ/2-geo.dVoxelZ/2;
    S2.x      =S2.x+geo.sVoxelX/2-geo.dVoxelX/2;          S2.y      =S2.y+geo.sVoxelY/2-geo.dVoxelY/2;              S2.z      =S2.z      +geo.sVoxelZ/2-geo.dVoxelZ/2;
    
    //4. Scale everything so dVoxel==1
    Pfinal.x  =Pfinal.x/geo.dVoxelX;      Pfinal.y  =Pfinal.y/geo.dVoxelY;        Pfinal.z  =Pfinal.z/geo.dVoxelZ;
    Pfinalu0.x=Pfinalu0.x/geo.dVoxelX;    Pfinalu0.y=Pfinalu0.y/geo.dVoxelY;      Pfinalu0.z=Pfinalu0.z/geo.dVoxelZ;
    Pfinalv0.x=Pfinalv0.x/geo.dVoxelX;    Pfinalv0.y=Pfinalv0.y/geo.dVoxelY;      Pfinalv0.z=Pfinalv0.z/geo.dVoxelZ;
    S2.x      =S2.x/geo.dVoxelX;          S2.y      =S2.y/geo.dVoxelY;            S2.z      =S2.z/geo.dVoxelZ;
    
    
      
    //5. apply COR. Wherever everything was, now its offesetd by a bit
    float CORx, CORy;
    CORx=-geo.COR[i]*sin(geo.alpha)/geo.dVoxelX;
    CORy= geo.COR[i]*cos(geo.alpha)/geo.dVoxelY;
    Pfinal.x+=CORx;   Pfinal.y+=CORy;
    Pfinalu0.x+=CORx;   Pfinalu0.y+=CORy;
    Pfinalv0.x+=CORx;   Pfinalv0.y+=CORy;
    S2.x+=CORx; S2.y+=CORy;
    
    // return
    
    *uvorigin=Pfinal;
    
    deltaU->x=Pfinalu0.x-Pfinal.x;
    deltaU->y=Pfinalu0.y-Pfinal.y;
    deltaU->z=Pfinalu0.z-Pfinal.z;
    
    deltaV->x=Pfinalv0.x-Pfinal.x;
    deltaV->y=Pfinalv0.y-Pfinal.y;
    deltaV->z=Pfinalv0.z-Pfinal.z;
    
    *source=S2;
}
