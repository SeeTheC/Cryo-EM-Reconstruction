#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA functions for texture-memory interpolation based projection
 *
 * This file has the necesary functions to perform X-ray parallel projection 
 * operation given a geaometry, angles and image. It uses the 3D texture 
 * memory linear interpolation to uniformily sample a path to integrate the 
 * X-rays.
 *
 * CODE by       Ander Biguri
 *               Sepideh Hatamikia (arbitrary rotation)
---------------------------------------------------------------------------
---------------------------------------------------------------------------
Copyright (c) 2015, University of Bath and CERN- European Organization for 
Nuclear Research
All rights reserved.

Redistribution and use in source and binary forms, with or without 
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, 
this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice, 
this list of conditions and the following disclaimer in the documentation 
and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors
may be used to endorse or promote products derived from this software without
specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" 
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE 
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE 
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE 
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR 
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF 
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN 
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.
 ---------------------------------------------------------------------------

Contact: tigre.toolbox@gmail.com
Codes  : https://github.com/CERN/TIGRE
--------------------------------------------------------------------------- 
 */



#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "ray_interpolated_projection_parallel.hpp"
#include "mex.h"
#include <math.h>

#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                mexErrMsgIdAndTxt("CBCT:CUDA:Atb",hipGetErrorString(__err));\
        } \
} while (0)
    
    
// Declare the texture reference.
    texture<float, hipTextureType3D , hipReadModeElementType> tex;

#define MAXTREADS 1024
/*GEOMETRY DEFINITION
 *
 *                Detector plane, behind
 *            |-----------------------------|
 *            |                             |
 *            |                             |
 *            |                             |
 *            |                             |
 *            |      +--------+             |
 *            |     /        /|             |
 *   A Z      |    /        / |*D           |
 *   |        |   +--------+  |             |
 *   |        |   |        |  |             |
 *   |        |   |     *O |  +             |
 *    --->y   |   |        | /              |
 *  /         |   |        |/               |
 * V X        |   +--------+                |
 *            |-----------------------------|
 *
 *           *S
 *
 *
 *
 *
 *
 **/


__global__ void kernelPixelDetector_parallel( Geometry geo,
        float* detector,
        Point3D source ,
        Point3D deltaU,
        Point3D deltaV,
        Point3D uvOrigin,
        float DSO,
        float maxdist){
    
    unsigned long y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned long x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long idx =  x  * geo.nDetecV + y;

    if ((x>= geo.nDetecU) | (y>= geo.nDetecV))
        return;
    
    

    
    /////// Get coordinates XYZ of pixel UV
    int pixelV = geo.nDetecV-y-1;
    int pixelU = x;
    
    
    
    float vectX,vectY,vectZ;
    Point3D P;
    P.x=(uvOrigin.x+pixelU*deltaU.x+pixelV*deltaV.x);
    P.y=(uvOrigin.y+pixelU*deltaU.y+pixelV*deltaV.y);
    P.z=(uvOrigin.z+pixelU*deltaU.z+pixelV*deltaV.z);
    Point3D S;
    S.x=(source.x+pixelU*deltaU.x+pixelV*deltaV.x);
    S.y=(source.y+pixelU*deltaU.y+pixelV*deltaV.y);
    S.z=(source.z+pixelU*deltaU.z+pixelV*deltaV.z);
    
    // Length is the ray length in normalized space
    double length=sqrt((S.x-P.x)*(S.x-P.x)+(S.y-P.y)*(S.y-P.y)+(S.z-P.z)*(S.z-P.z));
    //now legth is an integer of Nsamples that are required on this line
    length=ceil(length/geo.accuracy);//Divide the directional vector by an integer
    vectX=(P.x -S.x)/(length);
    vectY=(P.y -S.y)/(length);
    vectZ=(P.z -S.z)/(length);
    
    
//     //Integrate over the line
    float tx,ty,tz;
    float sum=0;
    float i;
    
    
    // limit the amount of mem access after the cube, but before the detector.
    if ((2*DSO/geo.dVoxelX+maxdist)/geo.accuracy  <   length)
        length=ceil((2*DSO/geo.dVoxelX+maxdist)/geo.accuracy);  
    //Length is not actually a length, but the amount of memreads with given accuracy ("samples per voxel")
    
    for (i=floor(maxdist/geo.accuracy); i<=length; i=i+1){
        tx=vectX*i+S.x;
        ty=vectY*i+S.y;
        tz=vectZ*i+S.z;
        
        sum += tex3D(tex, tx+0.5, ty+0.5, tz+0.5); // this line is 94% of time.
        
    }
    float deltalength=sqrt((vectX*geo.dVoxelX)*(vectX*geo.dVoxelX)+
            (vectY*geo.dVoxelY)*(vectY*geo.dVoxelY)+(vectZ*geo.dVoxelZ)*(vectZ*geo.dVoxelZ) );
    detector[idx]=sum*deltalength;
}



int interpolation_projection_parallel(float const * const img, Geometry geo, float** result,float const * const angles,int nangles){
    
 
    // copy data to CUDA memory

    hipArray *d_imagedata = 0;
    
    const hipExtent extent = make_hipExtent(geo.nVoxelX, geo.nVoxelY, geo.nVoxelZ);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipMalloc3DArray(&d_imagedata, &channelDesc, extent);
    cudaCheckErrors("hipMalloc3D error 3D tex");
    
    hipMemcpy3DParms copyParams = { 0 };
    copyParams.srcPtr = make_hipPitchedPtr((void*)img, extent.width*sizeof(float), extent.width, extent.height);
    copyParams.dstArray = d_imagedata;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);
    
    cudaCheckErrors("hipMemcpy3D fail");
    
    // Configure texture options
    tex.normalized = false;
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeBorder;
    tex.addressMode[1] = hipAddressModeBorder;
    tex.addressMode[2] = hipAddressModeBorder;
    
    hipBindTextureToArray(tex, d_imagedata, channelDesc);
    
    cudaCheckErrors("3D texture memory bind fail");
    
    
    //Done! Image put into texture memory.
    
    
    size_t num_bytes = geo.nDetecU*geo.nDetecV * sizeof(float);
    float* dProjection;
    hipMalloc((void**)&dProjection, num_bytes);
    cudaCheckErrors("hipMalloc fail");

    
//     If we are going to time
    bool timekernel=false;
    hipEvent_t start, stop;
    float elapsedTime;
    if (timekernel){
        hipEventCreate(&start);
        hipEventRecord(start,0);
    } 
    
    // 16x16 gave the best performance empirically
    // Funnily that makes it compatible with most GPUs.....
    dim3 grid(ceil((float)geo.nDetecU/32),ceil((float)geo.nDetecV/32),1);
    dim3 block(32,32,1); 
    Point3D source, deltaU, deltaV, uvOrigin;
    float maxdist;
    for (unsigned int i=0;i<nangles;i++){
        
        geo.alpha=angles[i*3];
        geo.theta=angles[i*3+1];
        geo.psi  =angles[i*3+2];
        //precomute distances for faster execution
        maxdist=maxdistanceCuboid(geo,i);
        //Precompute per angle constant stuff for speed
        computeDeltas_parallel(geo,geo.alpha,i, &uvOrigin, &deltaU, &deltaV, &source);
        //Interpolation!!
        
        kernelPixelDetector_parallel<<<grid,block>>>(geo,dProjection, source, deltaU, deltaV, uvOrigin,geo.DSO[i],floor(maxdist));
        cudaCheckErrors("Kernel fail");
        // copy result to host
        hipMemcpy(result[i], dProjection, num_bytes, hipMemcpyDeviceToHost);
        cudaCheckErrors("hipMemcpy fail");
        
           

    }
    if (timekernel){
        hipEventCreate(&stop);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start,stop);
        mexPrintf("%f\n" ,elapsedTime);
    }

    hipUnbindTexture(tex);
    cudaCheckErrors("Unbind  fail");
    
    hipFree(dProjection);
    hipFreeArray(d_imagedata);
    cudaCheckErrors("hipFree d_imagedata fail");
    
    
    
    hipDeviceReset();
    
    return 0;
}




/* This code precomputes The location of the source and the Delta U and delta V (in the warped space)
 * to compute the locations of the x-rays. While it seems verbose and overly-optimized,
 * it does saves about 30% of each of the kernel calls. Thats something!
 **/
void computeDeltas_parallel(Geometry geo, float alpha,unsigned int i, Point3D* uvorigin, Point3D* deltaU, Point3D* deltaV, Point3D* source){
    Point3D S;
    S.x=geo.DSO[i];
    S.y=geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);
    S.z=geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    
    //End point
    Point3D P,Pu0,Pv0;
    
    P.x  =-(geo.DSD[i]-geo.DSO[i]);   P.y  = geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       P.z  = geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pu0.x=-(geo.DSD[i]-geo.DSO[i]);   Pu0.y= geo.dDetecU*(1-((float)geo.nDetecU/2)+0.5);       Pu0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pv0.x=-(geo.DSD[i]-geo.DSO[i]);   Pv0.y= geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       Pv0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-1);
    // Geomtric trasnformations:
    
    //1: Offset detector
    
    //P.x
    P.y  =P.y  +geo.offDetecU[i];    P.z  =P.z  +geo.offDetecV[i];
    Pu0.y=Pu0.y+geo.offDetecU[i];    Pu0.z=Pu0.z+geo.offDetecV[i];
    Pv0.y=Pv0.y+geo.offDetecU[i];    Pv0.z=Pv0.z+geo.offDetecV[i];
    //S doesnt need to chagne
    
    
    //3: Rotate (around z)!
    Point3D Pfinal, Pfinalu0, Pfinalv0;
    Pfinal.x  =P.x;
    Pfinal.y  =P.y  +geo.offDetecU[i]; Pfinal.z  =P.z  +geo.offDetecV[i];
    Pfinalu0.x=Pu0.x;
    Pfinalu0.y=Pu0.y  +geo.offDetecU[i]; Pfinalu0.z  =Pu0.z  +geo.offDetecV[i];
    Pfinalv0.x=Pv0.x;
    Pfinalv0.y=Pv0.y  +geo.offDetecU[i]; Pfinalv0.z  =Pv0.z  +geo.offDetecV[i];
    
    eulerZYZ(geo,&Pfinal);
    eulerZYZ(geo,&Pfinalu0);
    eulerZYZ(geo,&Pfinalv0);
    eulerZYZ(geo,&S);
       
   
    
    //2: Offset image (instead of offseting image, -offset everything else)
    
    Pfinal.x  =Pfinal.x-geo.offOrigX[i];     Pfinal.y  =Pfinal.y-geo.offOrigY[i];     Pfinal.z  =Pfinal.z-geo.offOrigZ[i];
    Pfinalu0.x=Pfinalu0.x-geo.offOrigX[i];   Pfinalu0.y=Pfinalu0.y-geo.offOrigY[i];   Pfinalu0.z=Pfinalu0.z-geo.offOrigZ[i];
    Pfinalv0.x=Pfinalv0.x-geo.offOrigX[i];   Pfinalv0.y=Pfinalv0.y-geo.offOrigY[i];   Pfinalv0.z=Pfinalv0.z-geo.offOrigZ[i];
    S.x=S.x-geo.offOrigX[i];       S.y=S.y-geo.offOrigY[i];       S.z=S.z-geo.offOrigZ[i];
    
    // As we want the (0,0,0) to be in a corner of the image, we need to translate everything (after rotation);
    Pfinal.x  =Pfinal.x+geo.sVoxelX/2-geo.dVoxelX/2;      Pfinal.y  =Pfinal.y+geo.sVoxelY/2-geo.dVoxelY/2;          Pfinal.z  =Pfinal.z  +geo.sVoxelZ/2-geo.dVoxelZ/2;
    Pfinalu0.x=Pfinalu0.x+geo.sVoxelX/2-geo.dVoxelX/2;    Pfinalu0.y=Pfinalu0.y+geo.sVoxelY/2-geo.dVoxelY/2;        Pfinalu0.z=Pfinalu0.z+geo.sVoxelZ/2-geo.dVoxelZ/2;
    Pfinalv0.x=Pfinalv0.x+geo.sVoxelX/2-geo.dVoxelX/2;    Pfinalv0.y=Pfinalv0.y+geo.sVoxelY/2-geo.dVoxelY/2;        Pfinalv0.z=Pfinalv0.z+geo.sVoxelZ/2-geo.dVoxelZ/2;
    S.x       =S.x+geo.sVoxelX/2-geo.dVoxelX/2;           S.y       =S.y+geo.sVoxelY/2-geo.dVoxelY/2;               S.z       =S.z      +geo.sVoxelZ/2-geo.dVoxelZ/2;
    
    //4. Scale everything so dVoxel==1
    Pfinal.x  =Pfinal.x/geo.dVoxelX;      Pfinal.y  =Pfinal.y/geo.dVoxelY;        Pfinal.z  =Pfinal.z/geo.dVoxelZ;
    Pfinalu0.x=Pfinalu0.x/geo.dVoxelX;    Pfinalu0.y=Pfinalu0.y/geo.dVoxelY;      Pfinalu0.z=Pfinalu0.z/geo.dVoxelZ;
    Pfinalv0.x=Pfinalv0.x/geo.dVoxelX;    Pfinalv0.y=Pfinalv0.y/geo.dVoxelY;      Pfinalv0.z=Pfinalv0.z/geo.dVoxelZ;
    S.x       =S.x/geo.dVoxelX;           S.y       =S.y/geo.dVoxelY;             S.z       =S.z/geo.dVoxelZ;
    
    
      
    //5. apply COR. Wherever everything was, now its offesetd by a bit
    float CORx, CORy;
    CORx=-geo.COR[i]*sin(geo.alpha)/geo.dVoxelX;
    CORy= geo.COR[i]*cos(geo.alpha)/geo.dVoxelY;
    Pfinal.x+=CORx;   Pfinal.y+=CORy;
    Pfinalu0.x+=CORx;   Pfinalu0.y+=CORy;
    Pfinalv0.x+=CORx;   Pfinalv0.y+=CORy;
    S.x+=CORx; S.y+=CORy;
    
    // return
    
    *uvorigin=Pfinal;
    
    deltaU->x=Pfinalu0.x-Pfinal.x;
    deltaU->y=Pfinalu0.y-Pfinal.y;
    deltaU->z=Pfinalu0.z-Pfinal.z;
    
    deltaV->x=Pfinalv0.x-Pfinal.x;
    deltaV->y=Pfinalv0.y-Pfinal.y;
    deltaV->z=Pfinalv0.z-Pfinal.z;
    
    *source=S;
}
