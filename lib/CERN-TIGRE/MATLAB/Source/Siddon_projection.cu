#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA functions for ray-voxel intersection based projection
 *
 * This file has the necesary fucntiosn to perform X-ray CBCT projection
 * operation given a geaometry, angles and image. It usesthe so-called
 * Jacobs algorithm to compute efficiently the length of the x-rays over
 * voxel space.
 *
 * CODE by       Ander Biguri
 *               Sepideh Hatamikia (arbitrary rotation)
 * ---------------------------------------------------------------------------
 * ---------------------------------------------------------------------------
 * Copyright (c) 2015, University of Bath and CERN- European Organization for
 * Nuclear Research
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors
 * may be used to endorse or promote products derived from this software without
 * specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * ---------------------------------------------------------------------------
 *
 * Contact: tigre.toolbox@gmail.com
 * Codes  : https://github.com/CERN/TIGRE
 * ---------------------------------------------------------------------------
 */

#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "Siddon_projection.hpp"
#include "mex.h"
#include <math.h>

#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                mexErrMsgIdAndTxt("CBCT:CUDA:Atb",hipGetErrorString(__err));\
        } \
} while (0)
    
    
// Declare the texture reference.
    texture<float, hipTextureType3D , hipReadModeElementType> tex;
#define MAXTREADS 1024
/*GEOMETRY DEFINITION
 *
 *                Detector plane, behind
 *            |-----------------------------|
 *            |                             |
 *            |                             |
 *            |                             |
 *            |                             |
 *            |      +--------+             |
 *            |     /        /|             |
 *   A Z      |    /        / |*D           |
 *   |        |   +--------+  |             |
 *   |        |   |        |  |             |
 *   |        |   |     *O |  +             |
 *    --->y   |   |        | /              |
 *  /         |   |        |/               |
 * V X        |   +--------+                |
 *            |-----------------------------|
 *
 *           *S
 *
 *
 *
 *
 *
 **/



__global__ void kernelPixelDetector( Geometry geo,
        float* detector,
        Point3D source ,
        Point3D deltaU,
        Point3D deltaV,
        Point3D uvOrigin){
    
//     size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    unsigned long y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned long x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long idx =  x  * geo.nDetecV + y;
    
    if ((x>= geo.nDetecU) | (y>= geo.nDetecV))
        return;
    
    
    
    
    /////// Get coordinates XYZ of pixel UV
    int pixelV = geo.nDetecV-y-1;
    int pixelU = x;
    Point3D pixel1D;
    pixel1D.x=(uvOrigin.x+pixelU*deltaU.x+pixelV*deltaV.x);
    pixel1D.y=(uvOrigin.y+pixelU*deltaU.y+pixelV*deltaV.y);
    pixel1D.z=(uvOrigin.z+pixelU*deltaU.z+pixelV*deltaV.z);
    ///////
    // Siddon's ray-voxel intersection, optimized as in doi=10.1.1.55.7516
    //////
    // Also called Jacobs algorithms
    Point3D ray;
    // vector of Xray
    ray.x=pixel1D.x-source.x;
    ray.y=pixel1D.y-source.y;
    ray.z=pixel1D.z-source.z;
    // This variables are ommited because
    // bx,by,bz ={0,0,0}
    // dx,dy,dz ={1,1,1}
    // compute parameter values for x-ray parametric equation. eq(3-10)
    float axm,aym,azm;
    float axM,ayM,azM;
    // In the paper Nx= number of X planes-> Nvoxel+1
    axm=min(-source.x/ray.x,(geo.nVoxelX-source.x)/ray.x);
    aym=min(-source.y/ray.y,(geo.nVoxelY-source.y)/ray.y);
    azm=min(-source.z/ray.z,(geo.nVoxelZ-source.z)/ray.z);
    axM=max(-source.x/ray.x,(geo.nVoxelX-source.x)/ray.x);
    ayM=max(-source.y/ray.y,(geo.nVoxelY-source.y)/ray.y);
    azM=max(-source.z/ray.z,(geo.nVoxelZ-source.z)/ray.z);
    float am=max(max(axm,aym),azm);
    float aM=min(min(axM,ayM),azM);
    
    // line intersects voxel space ->   am<aM
    if (am>=aM)
        detector[idx]=0;
    
    // Compute max/min image INDEX for intersection eq(11-19)
    // Discussion about ternary operator in CUDA: https://stackoverflow.com/questions/7104384/in-cuda-why-is-a-b010-more-efficient-than-an-if-else-version
    float imin,imax,jmin,jmax,kmin,kmax;
    // for X
    if( source.x<pixel1D.x){
        imin=(am==axm)? 1             : ceil (source.x+am*ray.x);
        imax=(aM==axM)? geo.nVoxelX : floor(source.x+aM*ray.x);
    }else{
        imax=(am==axm)? geo.nVoxelX-1 : floor(source.x+am*ray.x);
        imin=(aM==axM)? 0             : ceil (source.x+aM*ray.x);
    }
    // for Y
    if( source.y<pixel1D.y){
        jmin=(am==aym)? 1             : ceil (source.y+am*ray.y);
        jmax=(aM==ayM)? geo.nVoxelY : floor(source.y+aM*ray.y);
    }else{
        jmax=(am==aym)? geo.nVoxelY-1 : floor(source.y+am*ray.y);
        jmin=(aM==ayM)? 0             : ceil (source.y+aM*ray.y);
    }
    // for Z
    if( source.z<pixel1D.z){
        kmin=(am==azm)? 1             : ceil (source.z+am*ray.z);
        kmax=(aM==azM)? geo.nVoxelZ : floor(source.z+aM*ray.z);
    }else{
        kmax=(am==azm)? geo.nVoxelZ-1 : floor(source.z+am*ray.z);
        kmin=(aM==azM)? 0             : ceil (source.z+aM*ray.z);
    }
    
    // get intersection point N1. eq(20-21) [(also eq 9-10)]
    float ax,ay,az;
    ax=(source.x<pixel1D.x)?  (imin-source.x)/(ray.x+0.000000000001) :  (imax-source.x)/(ray.x+0.000000000001);
    ay=(source.y<pixel1D.y)?  (jmin-source.y)/(ray.y+0.000000000001) :  (jmax-source.y)/(ray.y+0.000000000001);
    az=(source.z<pixel1D.z)?  (kmin-source.z)/(ray.z+0.000000000001) :  (kmax-source.z)/(ray.z+0.000000000001);
    
    
    
    // get index of first intersection. eq (26) and (19)
    int i,j,k;
    float aminc=min(min(ax,ay),az);
    i=(int)floor(source.x+ (aminc+am)/2*ray.x);
    j=(int)floor(source.y+ (aminc+am)/2*ray.y);
    k=(int)floor(source.z+ (aminc+am)/2*ray.z);
    // Initialize
    float ac=am;
    //eq (28), unit anlges
    float axu,ayu,azu;
    axu=1/abs(ray.x);
    ayu=1/abs(ray.y);
    azu=1/abs(ray.z);
    // eq(29), direction of update
    float iu,ju,ku;
    iu=(source.x< pixel1D.x)? 1 : -1;
    ju=(source.y< pixel1D.y)? 1 : -1;
    ku=(source.z< pixel1D.z)? 1 : -1;
    
    float maxlength=sqrt(ray.x*ray.x*geo.dVoxelX*geo.dVoxelX+ray.y*ray.y*geo.dVoxelY*geo.dVoxelY+ray.z*ray.z*geo.dVoxelZ*geo.dVoxelZ);
    float sum=0;
    unsigned int Np=(imax-imin+1)+(jmax-jmin+1)+(kmax-kmin+1); // Number of intersections
    // Go iterating over the line, intersection by intersection. If double point, no worries, 0 will be computed
    
    for (unsigned int ii=0;ii<Np;ii++){
        if (ax==aminc){
            sum+=(ax-ac)*tex3D(tex, i+0.5, j+0.5, k+0.5);
            i=i+iu;
            ac=ax;
            ax+=axu;
        }else if(ay==aminc){
            sum+=(ay-ac)*tex3D(tex, i+0.5, j+0.5, k+0.5);
            j=j+ju;
            ac=ay;
            ay+=ayu;
        }else if(az==aminc){
            sum+=(az-ac)*tex3D(tex, i+0.5, j+0.5, k+0.5);
            k=k+ku;
            ac=az;
            az+=azu;
        }
        aminc=min(min(ax,ay),az);
    }
    detector[idx]=sum*maxlength;
}


int siddon_ray_projection(float const * const img, Geometry geo, float** result,float const * const angles,int nangles){
    
    
    
    //DONE, Tesla found
    
    // copy data to CUDA memory
    hipArray *d_imagedata = 0;
    
    const hipExtent extent = make_hipExtent(geo.nVoxelX, geo.nVoxelY, geo.nVoxelZ);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipMalloc3DArray(&d_imagedata, &channelDesc, extent);
    cudaCheckErrors("hipMalloc3D error 3D tex");
    
    hipMemcpy3DParms copyParams = { 0 };
    copyParams.srcPtr = make_hipPitchedPtr((void*)img, extent.width*sizeof(float), extent.width, extent.height);
    copyParams.dstArray = d_imagedata;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);
    
    cudaCheckErrors("hipMemcpy3D fail");
    
    // Configure texture options
    tex.normalized = false;
    tex.filterMode = hipFilterModePoint; //we dont want interpolation
    tex.addressMode[0] = hipAddressModeBorder;
    tex.addressMode[1] = hipAddressModeBorder;
    tex.addressMode[2] = hipAddressModeBorder;
    
    hipBindTextureToArray(tex, d_imagedata, channelDesc);
    
    cudaCheckErrors("3D texture memory bind fail");
    
    
    
    
    //Done! Image put into texture memory.
    
    
    size_t num_bytes = geo.nDetecU*geo.nDetecV * sizeof(float);
    float* dProjection;
    hipMalloc((void**)&dProjection, num_bytes);
    hipMemset(dProjection,0,num_bytes);
    cudaCheckErrors("hipMalloc fail");
    
    
    bool timekernel=false; // For debuggin purposes
    hipEvent_t start, stop;
    float elapsedTime;

    Point3D source, deltaU, deltaV, uvOrigin;
    
    // 16x16 gave the best performance empirically
    // Funnily that makes it compatible with most GPUs.....
    int divU,divV;
    divU=16;
    divV=16;
    dim3 grid((geo.nDetecU+divU-1)/divU,(geo.nDetecV+divV-1)/divV,1);
    dim3 block(divU,divV,1);
    
    for (unsigned int i=0;i<nangles;i++){
        geo.alpha=angles[i*3];
        geo.theta=angles[i*3+1];
        geo.psi  =angles[i*3+2];
        //precomute distances for faster execution
        //Precompute per angle constant stuff for speed
        computeDeltas_Siddon(geo,i, &uvOrigin, &deltaU, &deltaV, &source);
        //Ray tracing!
        if (timekernel){
            hipEventCreate(&start);
            hipEventRecord(start,0);
        }
        kernelPixelDetector<<<grid,block>>>(geo,dProjection, source, deltaU, deltaV, uvOrigin);
        cudaCheckErrors("Kernel fail");
        if (timekernel){
            hipEventCreate(&stop);
            hipEventRecord(stop,0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsedTime, start,stop);
            mexPrintf("%f\n" ,elapsedTime);
        }
        // copy result to host
        hipMemcpy(result[i], dProjection, num_bytes, hipMemcpyDeviceToHost);
        cudaCheckErrors("hipMemcpy fail");
        
        
    }
    
    
    hipUnbindTexture(tex);
    cudaCheckErrors("Unbind  fail");
    hipFree(dProjection);
    hipFreeArray(d_imagedata);
    cudaCheckErrors("hipFree d_imagedata fail");
    
    
    
    hipDeviceReset();
    return 0;
}



/* This code precomputes The location of the source and the Delta U and delta V (in the warped space)
 * to compute the locations of the x-rays. While it seems verbose and overly-optimized,
 * it does saves about 30% of each of the kernel calls. Thats something!
 **/
void computeDeltas_Siddon(Geometry geo,int i, Point3D* uvorigin, Point3D* deltaU, Point3D* deltaV, Point3D* source){
    Point3D S;
    S.x=geo.DSO[i];
    S.y=0;
    S.z=0;
    
    //End point
    Point3D P,Pu0,Pv0;
    
    P.x  =-(geo.DSD[i]-geo.DSO[i]);   P.y  = geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       P.z  = geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pu0.x=-(geo.DSD[i]-geo.DSO[i]);   Pu0.y= geo.dDetecU*(1-((float)geo.nDetecU/2)+0.5);       Pu0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pv0.x=-(geo.DSD[i]-geo.DSO[i]);   Pv0.y= geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       Pv0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-1);
    // Geomtric trasnformations:
    // Now we have the Real world (OXYZ) coordinates of the bottom corner and its two neighbours.
    // The obkjective is to get a position of the detector in a coordinate system where:
    // 1-units are voxel size (in each direction can be different)
    // 2-The image has the its first voxel at (0,0,0)
    // 3-The image never rotates
    
    // To do that, we need to compute the "deltas" the detector, or "by how much
    // (in new xyz) does the voxels change when and index is added". To do that
    // several geometric steps needs to be changed
    
    //1.Roll,pitch,jaw
    // The detector can have a small rotation.
    // according to
    //"A geometric calibration method for cone beam CT systems" Yang K1, Kwan AL, Miller DF, Boone JM. Med Phys. 2006 Jun;33(6):1695-706.
    // Only the Z rotation will have a big influence in the image quality when they are small.
    // Still all rotations are supported
    
    // To roll pitch jaw, the detector has to be in centered in OXYZ.
    P.x=0;Pu0.x=0;Pv0.x=0;
    
    // Roll pitch yaw
    rollPitchYaw(geo,i,&P);
    rollPitchYaw(geo,i,&Pu0);
    rollPitchYaw(geo,i,&Pv0);
    //Now ltes translate the points where they shoudl be:
    P.x=P.x-(geo.DSD[i]-geo.DSO[i]);
    Pu0.x=Pu0.x-(geo.DSD[i]-geo.DSO[i]);
    Pv0.x=Pv0.x-(geo.DSD[i]-geo.DSO[i]);
    
    //1: Offset detector
    
   
    //S doesnt need to chagne
    
    
    //3: Rotate (around z)!
    Point3D Pfinal, Pfinalu0, Pfinalv0;
    Pfinal.x  =P.x;
    Pfinal.y  =P.y  +geo.offDetecU[i]; Pfinal.z  =P.z  +geo.offDetecV[i];
    Pfinalu0.x=Pu0.x;
    Pfinalu0.y=Pu0.y  +geo.offDetecU[i]; Pfinalu0.z  =Pu0.z  +geo.offDetecV[i];
    Pfinalv0.x=Pv0.x;
    Pfinalv0.y=Pv0.y  +geo.offDetecU[i]; Pfinalv0.z  =Pv0.z  +geo.offDetecV[i];
    
    eulerZYZ(geo,&Pfinal);
    eulerZYZ(geo,&Pfinalu0);
    eulerZYZ(geo,&Pfinalv0);
    eulerZYZ(geo,&S);
    
    //2: Offset image (instead of offseting image, -offset everything else)
    
    Pfinal.x  =Pfinal.x-geo.offOrigX[i];     Pfinal.y  =Pfinal.y-geo.offOrigY[i];     Pfinal.z  =Pfinal.z-geo.offOrigZ[i];
    Pfinalu0.x=Pfinalu0.x-geo.offOrigX[i];   Pfinalu0.y=Pfinalu0.y-geo.offOrigY[i];   Pfinalu0.z=Pfinalu0.z-geo.offOrigZ[i];
    Pfinalv0.x=Pfinalv0.x-geo.offOrigX[i];   Pfinalv0.y=Pfinalv0.y-geo.offOrigY[i];   Pfinalv0.z=Pfinalv0.z-geo.offOrigZ[i];
    S.x=S.x-geo.offOrigX[i];               S.y=S.y-geo.offOrigY[i];               S.z=S.z-geo.offOrigZ[i];
    
    // As we want the (0,0,0) to be in a corner of the image, we need to translate everything (after rotation);
    Pfinal.x  =Pfinal.x+geo.sVoxelX/2;      Pfinal.y  =Pfinal.y+geo.sVoxelY/2;          Pfinal.z  =Pfinal.z  +geo.sVoxelZ/2;
    Pfinalu0.x=Pfinalu0.x+geo.sVoxelX/2;    Pfinalu0.y=Pfinalu0.y+geo.sVoxelY/2;        Pfinalu0.z=Pfinalu0.z+geo.sVoxelZ/2;
    Pfinalv0.x=Pfinalv0.x+geo.sVoxelX/2;    Pfinalv0.y=Pfinalv0.y+geo.sVoxelY/2;        Pfinalv0.z=Pfinalv0.z+geo.sVoxelZ/2;
    S.x      =S.x+geo.sVoxelX/2;          S.y      =S.y+geo.sVoxelY/2;              S.z      =S.z      +geo.sVoxelZ/2;
    
    //4. Scale everything so dVoxel==1
    Pfinal.x  =Pfinal.x/geo.dVoxelX;      Pfinal.y  =Pfinal.y/geo.dVoxelY;        Pfinal.z  =Pfinal.z/geo.dVoxelZ;
    Pfinalu0.x=Pfinalu0.x/geo.dVoxelX;    Pfinalu0.y=Pfinalu0.y/geo.dVoxelY;      Pfinalu0.z=Pfinalu0.z/geo.dVoxelZ;
    Pfinalv0.x=Pfinalv0.x/geo.dVoxelX;    Pfinalv0.y=Pfinalv0.y/geo.dVoxelY;      Pfinalv0.z=Pfinalv0.z/geo.dVoxelZ;
    S.x      =S.x/geo.dVoxelX;          S.y      =S.y/geo.dVoxelY;            S.z      =S.z/geo.dVoxelZ;
    
    
    //mexPrintf("COR: %f \n",geo.COR[i]);
    //5. apply COR. Wherever everything was, now its offesetd by a bit
    float CORx, CORy;
    CORx=-geo.COR[i]*sin(geo.alpha)/geo.dVoxelX;
    CORy= geo.COR[i]*cos(geo.alpha)/geo.dVoxelY;
    Pfinal.x+=CORx;   Pfinal.y+=CORy;
    Pfinalu0.x+=CORx;   Pfinalu0.y+=CORy;
    Pfinalv0.x+=CORx;   Pfinalv0.y+=CORy;
    S.x+=CORx; S.y+=CORy;
    
    // return
    
    *uvorigin=Pfinal;
    
    deltaU->x=Pfinalu0.x-Pfinal.x;
    deltaU->y=Pfinalu0.y-Pfinal.y;
    deltaU->z=Pfinalu0.z-Pfinal.z;
    
    deltaV->x=Pfinalv0.x-Pfinal.x;
    deltaV->y=Pfinalv0.y-Pfinal.y;
    deltaV->z=Pfinalv0.z-Pfinal.z;
    
    *source=S;
}


#ifndef PROJECTION_HPP

float maxDistanceCubeXY(Geometry geo, float alpha,int i){
    ///////////
    // Compute initial "t" so we access safely as less as out of bounds as possible.
    //////////
    
    
    float maxCubX,maxCubY;
    // Forgetting Z, compute max distance: diagonal+offset
    maxCubX=(geo.sVoxelX/2+ abs(geo.offOrigX[i]))/geo.dVoxelX;
    maxCubY=(geo.sVoxelY/2+ abs(geo.offOrigY[i]))/geo.dVoxelY;
    
    return geo.DSO[i]/geo.dVoxelX-sqrt(maxCubX*maxCubX+maxCubY*maxCubY);
    
}
void rollPitchYaw(Geometry geo,int i, Point3D* point){
    Point3D auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x=cos(geo.dRoll[i])*cos(geo.dPitch[i])*auxPoint.x
            +(cos(geo.dRoll[i])*sin(geo.dPitch[i])*sin(geo.dYaw[i]) - sin(geo.dRoll[i])*cos(geo.dYaw[i]))*auxPoint.y
            +(cos(geo.dRoll[i])*sin(geo.dPitch[i])*cos(geo.dYaw[i]) + sin(geo.dRoll[i])*sin(geo.dYaw[i]))*auxPoint.z;
    
    point->y=sin(geo.dRoll[i])*cos(geo.dPitch[i])*auxPoint.x
            +(sin(geo.dRoll[i])*sin(geo.dPitch[i])*sin(geo.dYaw[i]) + cos(geo.dRoll[i])*cos(geo.dYaw[i]))*auxPoint.y
            +(sin(geo.dRoll[i])*sin(geo.dPitch[i])*cos(geo.dYaw[i]) - cos(geo.dRoll[i])*sin(geo.dYaw[i]))*auxPoint.z;
    
    point->z=-sin(geo.dPitch[i])*auxPoint.x
            +cos(geo.dPitch[1])*sin(geo.dYaw[i])*auxPoint.y
            +cos(geo.dPitch[1])*cos(geo.dYaw[i])*auxPoint.z;
    
}
void eulerZYZ(Geometry geo, Point3D* point){
    Point3D auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x=(+cos(geo.alpha)*cos(geo.theta)*cos(geo.psi)-sin(geo.alpha)*sin(geo.psi))*auxPoint.x+
             (-cos(geo.alpha)*cos(geo.theta)*sin(geo.psi)-sin(geo.alpha)*cos(geo.psi))*auxPoint.y+
              cos(geo.alpha)*sin(geo.theta)*auxPoint.z;
    
    point->y=(+sin(geo.alpha)*cos(geo.theta)*cos(geo.psi)+cos(geo.alpha)*sin(geo.psi))*auxPoint.x+
             (-sin(geo.alpha)*cos(geo.theta)*sin(geo.psi)+cos(geo.alpha)*cos(geo.psi))*auxPoint.y+
              sin(geo.alpha)*sin(geo.theta)*auxPoint.z;
    
    point->z=-sin(geo.theta)*cos(geo.psi)*auxPoint.x+
              sin(geo.theta)*sin(geo.psi)*auxPoint.y+
              cos(geo.theta)*auxPoint.z;


}
#endif
