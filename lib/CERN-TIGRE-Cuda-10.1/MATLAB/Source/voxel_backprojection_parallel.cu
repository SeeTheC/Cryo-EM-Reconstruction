#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA function for backrpojection  for parallel beam
 *
 *
 * CODE by  Ander Biguri
 *          Optimized and modified by RB
---------------------------------------------------------------------------
---------------------------------------------------------------------------
Copyright (c) 2015, University of Bath and CERN- European Organization for 
Nuclear Research
All rights reserved.

Redistribution and use in source and binary forms, with or without 
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, 
this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice, 
this list of conditions and the following disclaimer in the documentation 
and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its contributors
may be used to endorse or promote products derived from this software without
specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" 
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE 
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE 
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE 
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR 
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF 
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN 
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.
 ---------------------------------------------------------------------------

Contact: tigre.toolbox@gmail.com
Codes  : https://github.com/CERN/TIGRE
--------------------------------------------------------------------------- 
 */

 
#define  PI_2 1.57079632679489661923
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "voxel_backprojection.hpp"
#include "voxel_backprojection_parallel.hpp"

#include "mex.h"
#include <math.h>

// https://stackoverflow.com/questions/16282136/is-there-a-cuda-equivalent-of-perror
#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                mexErrMsgIdAndTxt("CBCT:CUDA:Atb",hipGetErrorString(__err));\
        } \
} while (0)
    
    
#define MAXTREADS 1024
    /*GEOMETRY DEFINITION
     *
     *                Detector plane, behind
     *            |-----------------------------|
     *            |                             |
     *            |                             |
     *            |                             |
     *            |                             |
     *            |      +--------+             |
     *            |     /        /|             |
     *   A Z      |    /        / |*D           |
     *   |        |   +--------+  |             |
     *   |        |   |        |  |             |
     *   |        |   |     *O |  +             |
     *   *--->y   |   |        | /              |
     *  /         |   |        |/               |
     * V X        |   +--------+                |
     *            |-----------------------------|
     *
     *           *S
     *
     *
     *
     *
     *
     **/
    texture<float, hipTextureType3D , hipReadModeElementType> tex;

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// RB, 10/31/2016: Add constant memory arrays to store parameters for all projections to be analyzed during a single kernel call
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// The optimal values of two constants obtained by RB on NVIDIA Quadro K2200 (4 GB RAM, 640 CUDA cores) for 512^3 volume and 512^3 projections (512 proj, each 512 x 512) were:
// PROJ_PER_KERNEL = 32 or 16 (very similar times)
// VOXELS_PER_THREAD = 8
// Speedup of the entire FDK backprojection (not only kernel run, also memcpy etc.) was nearly 4x relative to the original (single projection, single voxel per thread) code.
// (e.g. 16.2 s vs. ~62 s).

const int PROJ_PER_KERNEL = 32;  // Number of 2D projections to be analyzed by a single thread. This can be tweaked to see what works best. 32 was the optimal value in the paper by Zinsser and Keck.
const int VOXELS_PER_THREAD = 8;  // Number of voxels to be computed by s single thread. Can be tweaked to see what works best. 4 was the optimal value in the paper by Zinsser and Keck.

// We have PROJ_PER_KERNEL projections and we need 6 parameters for each projection:
//   deltaX, deltaY, deltaZ, xyzOrigin, offOrig, offDetec
// So we need to keep PROJ_PER_KERNEL*6 values in our deltas array FOR EACH CALL to our main kernel
// (they will be updated in the main loop before each kernel call).

__constant__ Point3D projParamsArrayDevParallel[6*PROJ_PER_KERNEL];  // Dev means it is on device

// We also need a corresponding array on the host side to be filled before each kernel call, then copied to the device (array in constant memory above)
Point3D projParamsArrayHostParallel[6*PROJ_PER_KERNEL];   // Host means it is host memory

// Now we also need to store sinAlpha and cosAlpha for each projection (two floats per projection)
__constant__ float projSinCosArrayDevParallel[3*PROJ_PER_KERNEL];

float projSinCosArrayHostParallel[3*PROJ_PER_KERNEL];

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// END RB, 10/31/2016: Add constant memory arrays to store parameters for all projections to be analyzed during a single kernel call
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////





//______________________________________________________________________________
//
//      Function:       rollPitchYaw
//
//      Description:    Main FDK backprojection kernel
//______________________________________________________________________________



//______________________________________________________________________________
//
//      Function:       kernelPixelBackprojectionFDK
//
//      Description:    Main FDK backprojection kernel
//______________________________________________________________________________

__global__ void kernelPixelBackprojection_parallel(const Geometry geo, float* image,const int currProjSetNumber, const int totalNoOfProjections)
{
    
    // Old kernel call signature:
    // kernelPixelBackprojectionFDK<<<grid,block>>>(geo,dimage,i,deltaX,deltaY,deltaZ,xyzOrigin,offOrig,offDetec,sinalpha,cosalpha);
    // We just read in most of the params from the constant memory instead of getting them from the param list.
    // This is because we now have MANY params, since single kernel processes more than one projection!
    /* __global__ void kernelPixelBackprojectionFDK(const Geometry geo,
     * float* image,
     * const int indAlpha,
     * const Point3D deltaX ,
     * const Point3D deltaY,
     * const Point3D deltaZ,
     * const Point3D xyzOrigin,
     * const Point3D xyzOffset,
     * const Point3D uv0Offset,
     * const float sinalpha,
     * const float cosalpha){
     */
    unsigned long indY = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned long indX = blockIdx.x * blockDim.x + threadIdx.x;
    // unsigned long startIndZ = blockIdx.z * blockDim.z + threadIdx.z;  // This is only STARTING z index of the column of voxels that the thread will handle
    unsigned long startIndZ = blockIdx.z * VOXELS_PER_THREAD + threadIdx.z;  // This is only STARTING z index of the column of voxels that the thread will handle
    //Make sure we dont go out of bounds
    if (indX>=geo.nVoxelX | indY>=geo.nVoxelY |startIndZ>=geo.nVoxelZ)
        return;
    
    // We'll keep a local auxiliary array of values of a column of voxels that this thread will update
    float voxelColumn[VOXELS_PER_THREAD];
    
    // First we need to copy the curent 3D volume values from the column to our auxiliary array so that we can then
    // work on them (update them by computing values from multiple projections) locally - avoiding main memory reads/writes
    
    int colIdx;
    
    for(colIdx=0; colIdx<VOXELS_PER_THREAD; colIdx++)
    {
        unsigned long indZ = startIndZ + colIdx;
        // If we are out of bounds, break the loop. The voxelColumn array will be updated partially, but it is OK, because we won't
        // be trying to copy the out of bounds values back to the 3D volume anyway (bounds checks will be done in the final loop where the updated values go back to the main volume)
        if(indZ>=geo.nVoxelZ)
            break;   // break the loop.
        
        unsigned long long idx =indZ*geo.nVoxelX*geo.nVoxelY+indY*geo.nVoxelX + indX;
        voxelColumn[colIdx] = image[idx];   // Read the current volume value that we'll update by computing values from MULTIPLE projections (not just one)
        // We'll be updating the local (register) variable, avoiding reads/writes from the slow main memory.
    }  // END copy 3D volume voxels to local array
    
    // Now iterate through projections
    for(int projNumber=0; projNumber<PROJ_PER_KERNEL; projNumber++)
    {
        // Get the current parameters from parameter arrays in constant memory.
        int indAlpha = currProjSetNumber*PROJ_PER_KERNEL+projNumber;  // This is the ABSOLUTE projection number in the projection array
        
        // Our currImageVal will be updated by hovewer many projections we had left in the "remainder" - that's OK.
        if(indAlpha>=totalNoOfProjections)
            break;
        
        Point3D deltaX = projParamsArrayDevParallel[6*projNumber];  // 6*projNumber because we have 6 Point3D values per projection
        Point3D deltaY = projParamsArrayDevParallel[6*projNumber+1];
        Point3D deltaZ = projParamsArrayDevParallel[6*projNumber+2];
        Point3D xyzOrigin = projParamsArrayDevParallel[6*projNumber+3];
        Point3D xyzOffset = projParamsArrayDevParallel[6*projNumber+4];
        Point3D S = projParamsArrayDevParallel[6*projNumber+5];
        
        float DSD = projSinCosArrayDevParallel[3*projNumber];     // 2*projNumber because we have 2 float (sin or cos angle) values per projection
        float DSO = projSinCosArrayDevParallel[3*projNumber+1];
        float COR = projSinCosArrayDevParallel[3*projNumber+2];

        // Geometric trasnformations:
        //Source, scaled XYZ coordinates
        
        // Now iterate through Z in our voxel column FOR A GIVEN PROJECTION
        for(colIdx=0; colIdx<VOXELS_PER_THREAD; colIdx++)
        {
            unsigned long indZ = startIndZ + colIdx;
            
            // If we are out of bounds, break the loop. The voxelColumn array will be updated partially, but it is OK, because we won't
            // be trying to copy the out of bounds values anyway (bounds checks will be done in the final loop where the values go to the main volume)
            if(indZ>=geo.nVoxelZ)
                break;   // break the loop.
            
            // "XYZ" in the scaled coordinate system of the current point. The image is rotated with the projection angles.
            Point3D P;
            S.x=DSO;
            P.x=(xyzOrigin.x+indX*deltaX.x+indY*deltaY.x+indZ*deltaZ.x);
            P.y=(xyzOrigin.y+indX*deltaX.y+indY*deltaY.y+indZ*deltaZ.y)-COR/geo.dDetecU;
            P.z=(xyzOrigin.z+indX*deltaX.z+indY*deltaY.z+indZ*deltaZ.z);
            S.y=P.y;S.z=P.z;

            // This is the vector defining the line from the source to the Voxel
            float vectX,vectY,vectZ;
            vectX=(P.x -S.x);
            vectY=(P.y -S.y);
            vectZ=(P.z -S.z);
            
            // Get the coordinates in the detector UV where the mid point of the voxel is projected.
            float t=(DSO-DSD /*-DOD*/ - S.x)/vectX;
            float y,z;
            y=vectY*t+S.y;
            z=vectZ*t+S.z;
            float u,v;
            u=y+geo.nDetecU/2-0.5;
            v=z+geo.nDetecV/2-0.5;
            
            
            
            // Get Value in the computed (U,V) and multiply by the corresponding weigth.
            // indAlpha is the ABSOLUTE number of projection in the projection array (NOT the current number of projection set!)
            voxelColumn[colIdx]+=tex3D(tex, v +0.5 ,
                    u +0.5 ,
                    indAlpha+0.5);

        }  // END iterating through column of voxels
        
    }  // END iterating through multiple projections
    
    // And finally copy the updated local voxelColumn array back to our 3D volume (main memory)
    for(colIdx=0; colIdx<VOXELS_PER_THREAD; colIdx++)
    {
        unsigned long indZ = startIndZ + colIdx;
        // If we are out of bounds, break the loop. The voxelColumn array will be updated partially, but it is OK, because we won't
        // be trying to copy the out of bounds values back to the 3D volume anyway (bounds checks will be done in the final loop where the values go to the main volume)
        if(indZ>=geo.nVoxelZ)
            break;   // break the loop.
        
        unsigned long long idx =indZ*geo.nVoxelX*geo.nVoxelY+indY*geo.nVoxelX + indX;
        image[idx] = voxelColumn[colIdx];   // Read the current volume value that we'll update by computing values from MULTIPLE projections (not just one)
        // We'll be updating the local (register) variable, avoiding reads/writes from the slow main memory.
        // According to references (Papenhausen), doing = is better than +=, since += requires main memory read followed by a write.
        // We did all the reads into the local array at the BEGINNING of this kernel. According to Papenhausen, this type of read-write split is
        // better for avoiding memory congestion.
    }  // END copy updated voxels from local array to our 3D volume
    
}  // END kernelPixelBackprojectionFDK




//______________________________________________________________________________
//
//      Function:       voxel_backprojection_parallel
//
//      Description:    Main host function for FDK backprojection (invokes the kernel)
//______________________________________________________________________________

int voxel_backprojection_parallel(float const * const projections, Geometry geo, float* result,float const * const alphas, int nalpha)
{
    
    /*
     * Allocate texture memory on the device
     */
    // copy data to CUDA memory
    hipArray *d_projectiondata = 0;
    const hipExtent extent = make_hipExtent(geo.nDetecV,geo.nDetecU,nalpha);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipMalloc3DArray(&d_projectiondata, &channelDesc, extent);
    cudaCheckErrors("hipMalloc3D error 3D tex");
    
    hipMemcpy3DParms copyParams = { 0 };
    copyParams.srcPtr = make_hipPitchedPtr((void*)projections, extent.width*sizeof(float), extent.width, extent.height);
    copyParams.dstArray = d_projectiondata;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);
    
    cudaCheckErrors("hipMemcpy3D fail");
    
    // Configure texture options
    tex.normalized = false;
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeBorder;
    tex.addressMode[1] = hipAddressModeBorder;
    tex.addressMode[2] = hipAddressModeBorder;
    
    hipBindTextureToArray(tex, d_projectiondata, channelDesc);
    
    cudaCheckErrors("3D texture memory bind fail");
    
    
    // Allocate result image memory
    size_t num_bytes = geo.nVoxelX*geo.nVoxelY*geo.nVoxelZ * sizeof(float);
    float* dimage;
    hipMalloc((void**)&dimage, num_bytes);
    hipMemset(dimage,0,num_bytes);
    cudaCheckErrors("hipMalloc fail");
    
    // If we are going to time
    bool timekernel=false;
    hipEvent_t start, stop;
    float elapsedTime;
    if (timekernel){
        hipEventCreate(&start);
        hipEventRecord(start,0);
    }
    
    int divx,divy,divz;
    
    // RB: Use the optimal (in their tests) block size from paper by Zinsser and Keck (16 in x and 32 in y).
    // I tried different sizes and shapes of blocks (tiles), but it does not appear to significantly affect trhoughput, so
    // let's stick with the values from Zinsser and Keck.
    divx=16;
    divy=32;
    divz=VOXELS_PER_THREAD;      // We now only have 32 x 16 threads per block (flat tile, see below), BUT each thread works on a Z column of VOXELS_PER_THREAD voxels, so we effectively need fewer blocks!
    dim3 grid((geo.nVoxelX+divx-1)/divx,
            (geo.nVoxelY+divy-1)/divy,
            (geo.nVoxelZ+divz-1)/divz);
    
    dim3 block(divx,divy,1);    // Note that we have 1 in the Z size, not divz, since each thread works on a vertical set of VOXELS_PER_THREAD voxels (so we only need a "flat" tile of threads, with depth of 1)
    
    //////////////////////////////////////////////////////////////////////////////////////
    // Main reconstruction loop: go through projections (rotation angles) and backproject
    //////////////////////////////////////////////////////////////////////////////////////
    
    // Since we'll have multiple projections processed by a SINGLE kernel call, compute how many
    // kernel calls we'll need altogether.
    int noOfKernelCalls = (nalpha+PROJ_PER_KERNEL-1)/PROJ_PER_KERNEL;  // We'll take care of bounds checking inside the loop if nalpha is not divisible by PROJ_PER_KERNEL
    for (unsigned int i=0; i<noOfKernelCalls; i++)
    {
        // Now we need to generate and copy all data for PROJ_PER_KERNEL projections to constant memory so that our kernel can use it
        int j;
        for(j=0; j<PROJ_PER_KERNEL; j++)
        {
            int currProjNumber=i*PROJ_PER_KERNEL+j;
            
            if(currProjNumber>=nalpha)
                break;  // Exit the loop. Even when we leave the param arrays only partially filled, this is OK, since the kernel will check bounds anyway.
            
            Point3D deltaX,deltaY,deltaZ,xyzOrigin, offOrig, /*offDetec,*/source;
            float sinalpha,cosalpha;
            
            geo.alpha=-alphas[currProjNumber*3];
//             sinalpha=sin(geo.alpha);
//             cosalpha=cos(geo.alpha);
            
            projSinCosArrayHostParallel[3*j]=geo.DSD[currProjNumber];  // 3*j because we have 3 float (sin or cos angle) values per projection
            projSinCosArrayHostParallel[3*j+1]=geo.DSO[currProjNumber];
            projSinCosArrayHostParallel[3*j+2]=geo.COR[currProjNumber];
            
            computeDeltasCubeParallel(geo,geo.alpha,currProjNumber,&xyzOrigin,&deltaX,&deltaY,&deltaZ);
            
            offOrig.x=geo.offOrigX[currProjNumber];
            offOrig.y=geo.offOrigY[currProjNumber];
            
            
            projParamsArrayHostParallel[6*j]=deltaX;		// 6*j because we have 6 Point3D values per projection
            projParamsArrayHostParallel[6*j+1]=deltaY;
            projParamsArrayHostParallel[6*j+2]=deltaZ;
            projParamsArrayHostParallel[6*j+3]=xyzOrigin;
            projParamsArrayHostParallel[6*j+4]=offOrig;
            projParamsArrayHostParallel[6*j+5]=source;
        }   // END for (preparing params for kernel call)
        
        // Copy the prepared parameter arrays to constant memory to make it available for the kernel
        hipMemcpyToSymbol(HIP_SYMBOL(projSinCosArrayDevParallel), projSinCosArrayHostParallel, sizeof(float)*3*PROJ_PER_KERNEL);
        hipMemcpyToSymbol(HIP_SYMBOL(projParamsArrayDevParallel), projParamsArrayHostParallel, sizeof(Point3D)*6*PROJ_PER_KERNEL);
        
        kernelPixelBackprojection_parallel<<<grid,block>>>(geo,dimage,i,nalpha);
        cudaCheckErrors("Kernel fail");
    }  // END for
    
    //////////////////////////////////////////////////////////////////////////////////////
    // END Main reconstruction loop: go through projections (rotation angles) and backproject
    //////////////////////////////////////////////////////////////////////////////////////
    
    
    if (timekernel)
    {
        hipEventCreate(&stop);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start,stop);
        mexPrintf("%f\n" ,elapsedTime);
        cudaCheckErrors("cuda Timing fail");
        
    }
    hipMemcpy(result, dimage, num_bytes, hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy result fail");
    
    hipUnbindTexture(tex);
    cudaCheckErrors("Unbind  fail");
    
    hipFree(dimage);
    hipFreeArray(d_projectiondata);
    cudaCheckErrors("hipFree d_imagedata fail");
    hipDeviceReset();
    return 0;
    
}  // END voxel_backprojection

void computeDeltasCubeParallel(Geometry geo, float alpha,int i, Point3D* xyzorigin, Point3D* deltaX, Point3D* deltaY, Point3D* deltaZ)
{
    Point3D P0, Px0,Py0,Pz0, source;
    // Get coords of Img(0,0,0)
    P0.x=-(geo.sVoxelX/2-geo.dVoxelX/2)+geo.offOrigX[i];
    P0.y=-(geo.sVoxelY/2-geo.dVoxelY/2)+geo.offOrigY[i];
    P0.z=-(geo.sVoxelZ/2-geo.dVoxelZ/2)+geo.offOrigZ[i];
    
    // Get coors from next voxel in each direction
    Px0.x=P0.x+geo.dVoxelX;       Py0.x=P0.x;                Pz0.x=P0.x;
    Px0.y=P0.y;                   Py0.y=P0.y+geo.dVoxelY;    Pz0.y=P0.y;
    Px0.z=P0.z;                   Py0.z=P0.z;                Pz0.z=P0.z+geo.dVoxelZ;
    
    // Rotate image (this is equivalent of rotating the source and detector)
    Point3D P, Px,Py,Pz; // We need other auxiliar variables to be able to perform the rotation, or we would overwrite values!
    P.x =P0.x *cos(alpha)-P0.y *sin(alpha);       P.y =P0.x *sin(alpha)+P0.y *cos(alpha);      P.z =P0.z;
    Px.x=Px0.x*cos(alpha)-Px0.y*sin(alpha);       Px.y=Px0.x*sin(alpha)+Px0.y*cos(alpha);      Px.z=Px0.z;
    Py.x=Py0.x*cos(alpha)-Py0.y*sin(alpha);       Py.y=Py0.x*sin(alpha)+Py0.y*cos(alpha);      Py.z=Py0.z;
    Pz.x=Pz0.x*cos(alpha)-Pz0.y*sin(alpha);       Pz.y=Pz0.x*sin(alpha)+Pz0.y*cos(alpha);      Pz.z=Pz0.z;
    
    //detector offset
    P.z =P.z-geo.offDetecV[i];          P.y =P.y-geo.offDetecU[i];
    Px.z =Px.z-geo.offDetecV[i];          Px.y =Px.y-geo.offDetecU[i];
    Py.z =Py.z-geo.offDetecV[i];          Py.y =Py.y-geo.offDetecU[i];
    Pz.z =Pz.z-geo.offDetecV[i];          Pz.y =Pz.y-geo.offDetecU[i];

    
//       mexPrintf("%f,%f,%f\n",source.x,source.y,source.z);
    // Scale coords so detector pixels are 1x1
    
    P.z =P.z /geo.dDetecV;                          P.y =P.y/geo.dDetecU;
    Px.z=Px.z/geo.dDetecV;                          Px.y=Px.y/geo.dDetecU;
    Py.z=Py.z/geo.dDetecV;                          Py.y=Py.y/geo.dDetecU;
    Pz.z=Pz.z/geo.dDetecV;                          Pz.y=Pz.y/geo.dDetecU;
    

    // get deltas of the changes in voxels
    deltaX->x=Px.x-P.x;   deltaX->y=Px.y-P.y;    deltaX->z=Px.z-P.z;
    deltaY->x=Py.x-P.x;   deltaY->y=Py.y-P.y;    deltaY->z=Py.z-P.z;
    deltaZ->x=Pz.x-P.x;   deltaZ->y=Pz.y-P.y;    deltaZ->z=Pz.z-P.z;
    
    
    *xyzorigin=P;
    
}  // END computeDeltasCube
