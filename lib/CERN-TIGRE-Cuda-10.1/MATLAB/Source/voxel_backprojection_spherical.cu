#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA function for backrpojection using FDK weigts for CBCT
 *
 *
 * CODE by  Ander Biguri & Sepideh Hatamikia
 * ---------------------------------------------------------------------------
 * ---------------------------------------------------------------------------
 * Copyright (c) 2015, University of Bath and CERN- European Organization for
 * Nuclear Research
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors
 * may be used to endorse or promote products derived from this software without
 * specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * ---------------------------------------------------------------------------
 *
 * Contact: tigre.toolbox@gmail.com
 * Codes  : https://github.com/CERN/TIGRE
 * ---------------------------------------------------------------------------
 */

#define  PI_2 1.57079632679489661923
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "voxel_backprojection.hpp"
#include "voxel_backprojection_spherical.hpp"
#include "mex.h"
#include <math.h>

// https://stackoverflow.com/questions/16282136/is-there-a-cuda-equivalent-of-perror
#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                mexErrMsgIdAndTxt("CBCT:CUDA:Atb",hipGetErrorString(__err));\
        } \
} while (0)
    
    
    
#define MAXTREADS 1024
    /*GEOMETRY DEFINITION
     *
     *                Detector plane, behind
     *            |-----------------------------|
     *            |                             |
     *            |                             |
     *            |                             |
     *            |                             |
     *            |      +--------+             |
              |     /        /|             |
     A Z      |    /        / |*D           |
     |        |   +--------+  |             |
     |        |   |        |  |             |
     |        |   |     *O |  +             |
     *--->y   |   |        | /              |
    /         |   |        |/               |
   V X        |   +--------+                |
     *            |-----------------------------|
     *
     *           *S
     *
     *
     *
     *
     *
     **/
    texture<float, hipTextureType3D , hipReadModeElementType> tex;


__global__ void FDKweigths(const Geometry geo,float* image,float constant){
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    for(; idx<geo.nVoxelX* geo.nVoxelY *geo.nVoxelZ; idx+=gridDim.x*blockDim.x) {
        image[idx]*=constant;
    }
    
}

__global__ void kernelPixelBackprojectionFDK_spherical(const Geometry geo,
        float* image,
        const int indAlpha,
        const float COR,
        const float DSD,
        const float DSO,
        const Point3D deltaX,
        const Point3D deltaY,
        const Point3D deltaZ,
        const Point3D xyzOrigin,
        const Point3D xyzOffset,
        const Point3D uv0Offset,
        const Point3D source){
    
    
    unsigned long indY = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned long indX = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long indZ = blockIdx.z * blockDim.z + threadIdx.z;
    //Make sure we dont go out of bounds
    unsigned long long idx =indZ*geo.nVoxelX*geo.nVoxelY+indY*geo.nVoxelX + indX;
    if (indX>=geo.nVoxelX | indY>=geo.nVoxelY |indZ>=geo.nVoxelZ)
        return;
    // Geometric trasnformations:
    
    //Source, scaled XYZ coordinates
    
    // "XYZ" in the scaled coordinate system of the current point. The iamge is rotated with the projection angles.
    Point3D P;

    P.x=(xyzOrigin.x+indX*deltaX.x+indY*deltaY.x+indZ*deltaZ.x);
    P.y=(xyzOrigin.y+indX*deltaX.y+indY*deltaY.y+indZ*deltaZ.y)-COR/geo.dDetecU;  
    P.z=(xyzOrigin.z+indX*deltaX.z+indY*deltaY.z+indZ*deltaZ.z);
    
    // This is the vector defining the line from the source to the Voxel
    float vectX,vectY,vectZ;
    vectX=(P.x -source.x);
    vectY=(P.y -source.y);
    vectZ=(P.z -source.z);
    
    
    // Get the coordinates in the detector UV where the mid point of the voxel is projected.
    float t=(DSO-DSD /*-DDO*/ - source.x)/vectX;
    float y,z;
    y=vectY*t+source.y;
    z=vectZ*t+source.z;
    float u,v;
    u=y+geo.nDetecU/2;
    v=z+geo.nDetecV/2;
    
    
    float weigth;
    float realx,realy;
    
    
    realx=-geo.sVoxelX/2+geo.dVoxelX/2    +indX*geo.dVoxelX   +xyzOffset.x;
    realy=-geo.sVoxelY/2+geo.dVoxelY/2    +indY*geo.dVoxelY   +xyzOffset.y+COR;
    
    
    weigth=(DSO+realy*sin(geo.alpha)-realx*cos(geo.alpha))/DSO; //TODO: This is wrong for shperical
    weigth=1/(weigth*weigth);
    
    // Get Value in the computed (U,V) and multiply by the corresponding weigth.
    image[idx]+=tex3D(tex, v ,
            u  ,
            indAlpha+0.5)
            *weigth;
//     image[idx]=v;
    
}


int voxel_backprojection_spherical(float const * const projections, Geometry geo, float* result,float const * const angles,int nalpha){
    
    
    /*
     * Allocate texture memory on the device
     */
    
    // copy data to CUDA memory
    hipArray *d_projectiondata = 0;
    const hipExtent extent = make_hipExtent(geo.nDetecV,geo.nDetecU,nalpha);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipMalloc3DArray(&d_projectiondata, &channelDesc, extent);
    cudaCheckErrors("hipMalloc3D error 3D tex");
    
    hipMemcpy3DParms copyParams = { 0 };
    copyParams.srcPtr = make_hipPitchedPtr((void*)projections, extent.width*sizeof(float), extent.width, extent.height);
    copyParams.dstArray = d_projectiondata;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);
    
    cudaCheckErrors("hipMemcpy3D fail");
    
    // Configure texture options
    tex.normalized = false;
    tex.filterMode = hipFilterModeLinear;
    tex.addressMode[0] = hipAddressModeBorder;
    tex.addressMode[1] = hipAddressModeBorder;
    tex.addressMode[2] = hipAddressModeBorder;
    
    hipBindTextureToArray(tex, d_projectiondata, channelDesc);
    
    cudaCheckErrors("3D texture memory bind fail");
    
    
    // Allocate result image memory
    size_t num_bytes = geo.nVoxelX*geo.nVoxelY*geo.nVoxelZ * sizeof(float);
    float* dimage;
    hipMalloc((void**)&dimage, num_bytes);
    hipMemset(dimage,0,num_bytes);
    cudaCheckErrors("hipMalloc fail");
    
    // If we are going to time
    bool timekernel=false;
    hipEvent_t start, stop;
    float elapsedTime;
    if (timekernel){
        hipEventCreate(&start);
        hipEventRecord(start,0);
    }
    
    int divx,divy,divz;
    
    //enpirical
    divx=32;
    divy=32;
    divz=1;
    dim3 grid((geo.nVoxelX+divx-1)/divx,
            (geo.nVoxelY+divy-1)/divy,
            (geo.nVoxelZ+divz-1)/divz);
    dim3 block(divx,divy,divz);
    Point3D deltaX,deltaY,deltaZ,xyzOrigin, offOrig, offDetec,source;
    for (unsigned int i=0;i<nalpha;i++){
        geo.alpha=-angles[i*3];
        geo.theta=-angles[i*3+1];
        geo.psi  =-angles[i*3+2];
        
        computeDeltasCubeSpherical(geo,i,&xyzOrigin,&deltaX,&deltaY,&deltaZ,&source);
        
        offOrig.x=geo.offOrigX[i];
        offOrig.y=geo.offOrigY[i];
        offDetec.x=geo.offDetecU[i];
        offDetec.y=geo.offDetecV[i];
        
        kernelPixelBackprojectionFDK_spherical<<<grid,block>>>(geo,dimage,i,geo.COR[i],geo.DSD[i],geo.DSO[i],deltaX,deltaY,deltaZ,xyzOrigin,offOrig,offDetec,source);
        cudaCheckErrors("Kernel fail");
    }
    if (timekernel){
        hipEventCreate(&stop);
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start,stop);
        mexPrintf("%f\n" ,elapsedTime);
        cudaCheckErrors("cuda Timing fail");
        
    }
    hipMemcpy(result, dimage, num_bytes, hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy result fail");
    
    hipUnbindTexture(tex);
    cudaCheckErrors("Unbind  fail");
    
    hipFree(dimage);
    hipFreeArray(d_projectiondata);
    cudaCheckErrors("hipFree d_imagedata fail");
    //hipDeviceReset();
    return 0;
    
}
void computeDeltasCubeSpherical(Geometry geo, int i, Point3D* xyzorigin, Point3D* deltaX, Point3D* deltaY, Point3D* deltaZ,Point3D *S){
    
    Point3D P, Px,Py,Pz;
    // Get coords of Img(0,0,0)
    P.x=-(geo.sVoxelX/2-geo.dVoxelX/2)+geo.offOrigX[i];
    P.y=-(geo.sVoxelY/2-geo.dVoxelY/2)+geo.offOrigY[i];
    P.z=-(geo.sVoxelZ/2-geo.dVoxelZ/2)+geo.offOrigZ[i];
    
    // Get coors from next voxel in each direction
    Px.x=P.x+geo.dVoxelX;       Py.x=P.x;                Pz.x=P.x;
    Px.y=P.y;                   Py.y=P.y+geo.dVoxelY;    Pz.y=P.y;
    Px.z=P.z;                   Py.z=P.z;                Pz.z=P.z+geo.dVoxelZ;
    
    
    
// Rotate image around X axis (this is equivalent of rotating the source and detector) RZ RY RZ
    
    eulerZYZT(geo,&P);
    eulerZYZT(geo,&Px);
    eulerZYZT(geo,&Py);
    eulerZYZT(geo,&Pz);

    
    
    //detector offset
    P.z =P.z-geo.offDetecV[i];            P.y =P.y-geo.offDetecU[i];
    Px.z =Px.z-geo.offDetecV[i];          Px.y =Px.y-geo.offDetecU[i];
    Py.z =Py.z-geo.offDetecV[i];          Py.y =Py.y-geo.offDetecU[i];
    Pz.z =Pz.z-geo.offDetecV[i];          Pz.y =Pz.y-geo.offDetecU[i];
    
    //Detector Roll pitch Yaw
    //
    //
    // first, we need to offset everything so (0,0,0) is the center of the detector
    // Only X is required for that
    P.x=P.x+(geo.DSD[i]-geo.DSO[i]);
    Px.x=Px.x+(geo.DSD[i]-geo.DSO[i]);
    Py.x=Py.x+(geo.DSD[i]-geo.DSO[i]);
    Pz.x=Pz.x+(geo.DSD[i]-geo.DSO[i]);
    rollPitchYawT(geo,i,&P);
    rollPitchYawT(geo,i,&Px);
    rollPitchYawT(geo,i,&Py);
    rollPitchYawT(geo,i,&Pz);
    
    P.x=P.x-(geo.DSD[i]-geo.DSO[i]);
    Px.x=Px.x-(geo.DSD[i]-geo.DSO[i]);
    Py.x=Py.x-(geo.DSD[i]-geo.DSO[i]);
    Pz.x=Pz.x-(geo.DSD[i]-geo.DSO[i]);
    //Done for P, now source
    Point3D source;
    source.x=geo.DSD[i]; //allready offseted for rotation
    source.y=-geo.offDetecU[i];
    source.z=-geo.offDetecV[i];
    rollPitchYawT(geo,i,&source);
    
    
    source.x=source.x-(geo.DSD[i]-geo.DSO[i]);//   source.y=source.y-auxOff.y;    source.z=source.z-auxOff.z;
    
//       mexPrintf("%f,%f,%f\n",source.x,source.y,source.z);
    // Scale coords so detector pixels are 1x1
    
    P.z =P.z /geo.dDetecV;                          P.y =P.y/geo.dDetecU;
    Px.z=Px.z/geo.dDetecV;                          Px.y=Px.y/geo.dDetecU;
    Py.z=Py.z/geo.dDetecV;                          Py.y=Py.y/geo.dDetecU;
    Pz.z=Pz.z/geo.dDetecV;                          Pz.y=Pz.y/geo.dDetecU;
    
    source.z=source.z/geo.dDetecV;                  source.y=source.y/geo.dDetecU;
    
    // get deltas of the changes in voxels
    deltaX->x=Px.x-P.x;   deltaX->y=Px.y-P.y;    deltaX->z=Px.z-P.z;
    deltaY->x=Py.x-P.x;   deltaY->y=Py.y-P.y;    deltaY->z=Py.z-P.z;
    deltaZ->x=Pz.x-P.x;   deltaZ->y=Pz.y-P.y;    deltaZ->z=Pz.z-P.z;
    
    
    *xyzorigin=P;
    *S=source;
}

void eulerZYZT(Geometry geo, Point3D* point){
    
    Point3D auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x = auxPoint.x*(cos(geo.psi)*cos(geo.theta)*cos(geo.alpha)-sin(geo.psi)*sin(geo.alpha)) 
              +auxPoint.y*(-cos(geo.psi)*cos(geo.theta)*sin(geo.alpha)-sin(geo.psi)*cos(geo.alpha))
              +auxPoint.z*cos(geo.psi)*sin(geo.theta);
    point->y = auxPoint.x*(sin(geo.psi)*cos(geo.theta)*cos(geo.alpha)+cos(geo.psi)*sin(geo.alpha))
              +auxPoint.y*(-sin(geo.psi)*cos(geo.theta)*sin(geo.alpha)+cos(geo.psi)*cos(geo.alpha))
              +auxPoint.z*sin(geo.psi)*sin(geo.theta);
    point->z =-auxPoint.x*sin(geo.theta)*cos(geo.alpha)
              +auxPoint.y*sin(geo.theta)*sin(geo.alpha)
              +auxPoint.z*cos(geo.theta);
}